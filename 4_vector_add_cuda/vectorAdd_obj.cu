
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

template <class T>
class vector3d
{
    public: 
        T x, y, z;

        vector3d()
        {
            x = 0;
            y = 0;
            z = 0;
        }

        vector3d(T x_, T y_, T z_)
        {
            x = x_;
            y = y_;
            z = z_;
        }

        inline vector3d& operator +=(const vector3d& rhs)
        {
            x += rhs.x;
            y += rhs.y;
            z += rhs.z;
            return *this;
        }

        inline vector3d& operator +(const vector3d& rhs)
        {
            *this += rhs;
            return *this;
        }

        inline vector3d& operator /=(const T& rhs)
        {
            x /= rhs;
            y /= rhs;
            z /= rhs;
            return *this;
        }

        inline vector3d& operator /(const T& rhs)
        {
            *this /= rhs;
            return *this;
        }

        __host__ __device__ inline void test()
        {
            int test = 0;
            test++;
        }
        
};

__global__ void vecAdd(vector3d<float> *a, vector3d<float> *b, vector3d<float> *c, int n)
{
    int id = blockIdx.x*blockDim.x+threadIdx.x;

    if (id < n)
    {
        c[id].x = a[id].x + b[id].x;
        c[id].y = a[id].y + b[id].y;
        c[id].z = a[id].z + b[id].z;

        c[id].test();
    }
}
 
int main( int argc, char* argv[] )
{
    // Size of vectors
    int n = 1000000;
    float n_f = 1000000.0f;
 		
    // Host input vectors
    vector3d<float> *h_a;
    vector3d<float> *h_b;
    
    //Host output vector
    vector3d<float> *h_c;
 
    // Device input vectors
    vector3d<float> *d_a;
    vector3d<float> *d_b;

    //Device output vector
    vector3d<float> *d_c;
 
    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(vector3d<float>);
 
    // Allocate memory for each vector on host
    h_a = new vector3d<float>[bytes](); //(vector3d<float>*)malloc(bytes);
    h_b = new vector3d<float>[bytes](); //(vector3d<float>*)malloc(bytes);
    h_c = new vector3d<float>[bytes](); //(vector3d<float>*)malloc(bytes);
 
    // Allocate memory for each vector on GPU
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
 
    // Initialize vectors on host
    for(int i = 0; i <= n; i++ ) {
        h_a[i] = vector3d<float>(1,2,3);
        h_b[i] = vector3d<float>(1,2,3);
    }
 
    // Copy host vectors to device
    hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);
 
    int blockSize, gridSize;
 
    // Number of threads in each thread block
    blockSize = 1024;
 
    // Number of thread blocks in grid
    gridSize = (int)ceil((float)n/blockSize);
 
    // Execute the kernel
    vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

    // Copy array back to host
    hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );
 
    // Sum up vector c and print result divided by n, this should equal 1 within error
    vector3d<float> sum;
    for(int i=0; i<=n; i++)
        sum += h_c[i];

    printf("final result: %f, %f, %f\n", sum.x / n_f, sum.y / n_f, sum.z / n_f);
    printf("final result: %f, %f, %f\n", sum.x, sum.y, sum.z);
 
    // Release device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
 
    // Release host memory
    free(h_a);
    free(h_b);
    free(h_c);
 
    return 0;
}
